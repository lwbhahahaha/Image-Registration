
#include <hip/hip_runtime.h>
__global__ void addToVector(float * pi, float c)  {
       // Location in a 2D matrix
       int idx = threadIdx.x+blockIdx.x*blockDim.x;
       int idy = threadIdx.y+blockIdx.y*blockDim.y;
       // The Location in a 2D matrix, defined by a 1D value
       int id  = idx+idy*(blockDim.x*gridDim.x); 
       pi[id] += c;
}
